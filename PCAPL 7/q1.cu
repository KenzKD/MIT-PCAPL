
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define MASK_WIDTH 5
#define WIDTH 10

/* 
do 1D conv
1 - with mask in global memory
2 - with mask in shared memory(tiled)
3 - with mask in constant memory
*/

__constant__ int d_Mc[MASK_WIDTH];

__global__ void conv_global_(int *A, int *M, int *R)
{
	
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    
	for(int j=0; j<MASK_WIDTH;j++)
		if(start+j>=0 && start+j<WIDTH)
			val+= A[start+j]*M[j];
	R[i]=val;
}

__global__ void conv_shared_(int *A, int *M, int *R)
{
    __shared__ int d_Ms[MASK_WIDTH];
    for(int i=0;i<MASK_WIDTH;i++)
        d_Ms[i] = M[i];
    
    int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    
	for(int j =0; j<MASK_WIDTH;j++)
		if(start+j>=0 && start+j<WIDTH)
			val+= A[start+j]*d_Ms[j];
	R[i]=val;
}

__global__ void conv_constant_(int *A, int *R)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
	float val = 0;
	int start = i-(MASK_WIDTH/2);
    
	for(int j =0; j<MASK_WIDTH;j++)
        if(start+j>=0 && start+j<WIDTH)
            val+= A[start+j]*d_Mc[j];
	R[i]=val;
}

int main()
{
    int A[WIDTH], M[MASK_WIDTH], R[WIDTH];
    int *d_A, *d_M, *d_R;

    for(int i=0; i<WIDTH; i++)
    {
        A[i] = rand() % 10;
        R[i] = 0;

        if(i < MASK_WIDTH)
            M[i] = rand() % 10;
    }

    printf("Input Matrix : ");
    for(int i=0; i<WIDTH; i++)
        printf("%d ", A[i]);

    printf("\nMask Matrix : ");
    for(int i=0; i<MASK_WIDTH; i++)
        printf("%d ", M[i]);

    hipMalloc((void**)&d_A, WIDTH*sizeof(int));
    hipMalloc((void**)&d_M, MASK_WIDTH*sizeof(int));
    hipMalloc((void**)&d_R, WIDTH*sizeof(int));

    hipMemcpy(d_A, A, WIDTH*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, MASK_WIDTH*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_R, R, WIDTH*sizeof(int), hipMemcpyHostToDevice);

    // Q1a
    conv_global_<<<1, WIDTH>>>(d_A, d_M, d_R);
    hipMemcpy(R, d_R, WIDTH*sizeof(int), hipMemcpyDeviceToHost);
    printf("\n\nResultant Matrix after Basic conv : ");
    for(int i=0; i<WIDTH; i++)
        printf("%d ", R[i]);

    // Q1b
    conv_shared_<<<1, WIDTH>>>(d_A, d_M, d_R);
    hipMemcpy(R, d_R, WIDTH*sizeof(int), hipMemcpyDeviceToHost);
    printf("\nResultant Matrix after Tiled conv : ");
    for(int i=0; i<WIDTH; i++)
        printf("%d ", R[i]);

    // Q1c
    hipMemcpyToSymbol(HIP_SYMBOL(d_Mc), M, MASK_WIDTH*sizeof(int));
    conv_constant_<<<1, WIDTH>>>(d_A, d_R);
    hipMemcpy(R, d_R, WIDTH*sizeof(int), hipMemcpyDeviceToHost);
    printf("\nResultant Matrix after constant conv : ");
    for(int i=0; i<WIDTH; i++)
        printf("%d ", R[i]);

    printf("\n");

    hipFree(d_A);
    hipFree(d_M);
    hipFree(d_R);
    return 0;
}