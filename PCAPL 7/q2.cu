#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"
#include""

__global__ void SpMV_CSR(int num_rows,int *data,int *col_index,int *row_ptr,int *x,int *y)
{
	int row=threadIdx.x;
	if(row<num_rows)
	{
		int dot=0;
		int row_start=row_ptr[row];
		int row_end=row_ptr[row+1];
		for(int i=row_start;i<row_end;i++)
			dot+= data[i]*x[col_index[i]];
		y[row]=dot;
	}
}
int main()
{
	int n=4;
	int y[n],row_ptr[n+1];
	int ipmat[n][n]={{0,0,3,4},{0,0,0,0},{0,5,0,7},{0,2,6,0}};
	int x[]={7,8,9,10};
	int nonzerocount=0;

	//finding number of non zero elements and row ptr array
	for(int i=0;i<n;i++)
	{
		row_ptr[i]=nonzerocount;
		for(int j=0;j<n;j++)
		{
			if(ipmat[i][j]!=0)
				nonzerocount++;
			printf("%d\t",ipmat[i][j]);
		}
		printf("\n");
	}

	row_ptr[n]=nonzerocount;
	int data[nonzerocount],col_index[nonzerocount];
	int k=0;
	//finding data and col_index array
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			if(ipmat[i][j]!=0)
			{
				data[k]=ipmat[i][j];
				col_index[k++]=j;
			}
		}
	}
	printf("\ndata array\t");
	for(int i=0;i<nonzerocount;i++)
		printf("%d\t",data[i]);
	
	printf("\ncol_index array\t");
	for(int i=0;i<nonzerocount;i++)
		printf("%d\t",col_index[i]);
	
	printf("\nrow_ptr array\t");
	for(int i=0;i<=n;i++)
		printf("%d\t",row_ptr[i]);
	
	printf("\nvector X\t");
	for(int i=0;i<n;i++)
		printf("%d\t",x[i]);
	
	int *d_data,*d_col_index,*d_row_ptr,*d_x,*d_y;

	//memory allocations
	hipMalloc((void**)&d_data,nonzerocount*sizeof(int));
	hipMalloc((void**)&d_col_index,nonzerocount*sizeof(int));
	hipMalloc((void**)&d_row_ptr,(n+1)*sizeof(int));
	hipMalloc((void**)&d_x,n*sizeof(int));
	hipMalloc((void**)&d_y,n*sizeof(int));

	//copy from host to device
	hipMemcpy(d_data,data,nonzerocount*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_col_index,col_index,nonzerocount*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr,row_ptr,(n+1)*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);

	//run kernel
	SpMV_CSR<<<1,n>>>(n,d_data,d_col_index,d_row_ptr,d_x,d_y);

	//copy from device to host
	hipMemcpy(y,d_y,n*sizeof(int),hipMemcpyDeviceToHost);
	printf("\nresult\t\t");
	for(int i=0;i<n;i++)
		printf("%d\t",y[i]);

	printf("\n");
	
	//free memory
	hipFree(d_data);
	hipFree(d_col_index);
	hipFree(d_row_ptr);
	hipFree(d_x);
	hipFree(d_y);
	return 0;
}