#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vecAddKernel_1ab(int* A, int* B, int* C)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	C[idx] = A[idx] + B[idx];
}

__global__ void vecAddKernel_1c(int* A, int* B, int* C, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < n)
		C[idx] = A[idx] + B[idx];
}

void vecAdd(int* A, int* B, int* C, int n)
{
	int size = n*sizeof(int);
	
	int *d_A,*d_B,*d_C;
	
	hipMalloc((void**) &d_A, size);
	hipMalloc((void**) &d_B, size);
	hipMalloc((void**) &d_C, size);
	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	
	printf("A: ");
	for (int i = 0; i < n; i++)
		printf("%d ", A[i]);
	
	printf("\nB: ");
	for (int i = 0; i < n; i++)
		printf("%d ", B[i]);
	
	// 1a
	vecAddKernel_1ab<<<n, 1>>>(d_A, d_B, d_C);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printf("\n\nA+B (1a): ");
	for (int i = 0; i < n ; i++)
		printf("%d ", C[i]);
	
	// 1b
	vecAddKernel_1ab<<<1, n>>>(d_A, d_B, d_C);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printf("\nA+B (1b): ");
	for (int i = 0; i < n ; i++)
		printf("%d ", C[i]);	

	// 1c
	vecAddKernel_1c<<<ceil(n/256.0), n>>>(d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	printf("\nA+B (1c): ");
	for (int i = 0; i < n ; i++)
		printf("%d ", C[i]);
	
	printf("\n");
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

int main()
{
	int *h_A, *h_B, *h_C;
	int n = 5;
	int size = n * sizeof(int);
	
	h_A = (int*) malloc(size);
	h_B = (int*) malloc(size);
	h_C = (int*) malloc(size);
	
	for (int i = 0; i < n; i++)
	{
		h_A[i] = i+1;
		h_B[i] = (i+1)*2;
	}

	vecAdd(h_A, h_B, h_C, n);
}