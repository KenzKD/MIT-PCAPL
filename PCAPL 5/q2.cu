#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void selectionSortKernel(int* unsorted, int* sorted, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int key = unsorted[idx];
	int pos = 0;
	
	for (int i=0;i<n;i++)
		if (unsorted[i]<key || (unsorted[i]==key && i<idx))
			pos++;

	sorted[pos] = key;
}

void selectionSort(int* unsorted, int* sorted, int n)
{
	int size = n*sizeof(int);
	int *d_unsorted,*d_sorted;
	
	hipMalloc((void**)&d_unsorted,size);
	hipMalloc((void**)&d_sorted,size);

	hipMemcpy(d_unsorted,unsorted,size,hipMemcpyHostToDevice);
	
	selectionSortKernel<<<1,n>>>(d_unsorted,d_sorted,n);
	
	hipMemcpy(sorted,d_sorted,size,hipMemcpyDeviceToHost);
	
	hipFree(d_unsorted);
	hipFree(d_sorted);
}

int main()
{
	int *h_unsorted,*h_sorted;
	int n = 5;
	int size = n*sizeof(int);
	
	h_unsorted = (int*)malloc(size);
	h_sorted = (int*)malloc(size);
	
	for (int i=0;i<5;i++)
		h_unsorted[i] = rand()%50;
	
	selectionSort(h_unsorted,h_sorted,n);
	
	printf("Unsorted Array: ");
	for (int i=0;i<n;i++)
		printf("%d ",h_unsorted[i]);
	
	printf("\nSorted Array: ");
	for (int i=0;i<n;i++)
		printf("%d ",h_sorted[i]);
	
	printf("\n");
}
