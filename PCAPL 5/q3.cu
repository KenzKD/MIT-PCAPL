#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void oddEven(int* arr, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx%2==1 && idx+1<n)
		if (arr[idx]>arr[idx+1])
		{
			int temp = arr[idx];
			arr[idx] = arr[idx+1];
			arr[idx+1] = temp;
		}
}

__global__ void evenOdd(int* arr, int n)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx%2==0 && idx+1<n)
		if (arr[idx]>arr[idx+1])
		{
			int temp = arr[idx];
			arr[idx] = arr[idx+1];
			arr[idx+1] = temp;
		}
}

void oddEvenTranspositionSort(int* arr, int n)
{
	int size = n*sizeof(int);
	int* d_arr;
	
	hipMalloc((void**) &d_arr, size);
	hipMemcpy(d_arr,arr,size,hipMemcpyHostToDevice);
	for (int i=0;i<=n/2;i++)
	{
		oddEven<<<1,n>>>(d_arr,n);
		evenOdd<<<1,n>>>(d_arr,n);
	}
	hipMemcpy(arr,d_arr,size,hipMemcpyDeviceToHost);
	hipFree(d_arr);
}

int main()
{
	int *h_arr;
	int n = 5;
	int size = n * sizeof(int);
	h_arr = (int*) malloc(size);
	
	printf("Unsorted Array: ");
	for (int i = 0; i < n; i++)
	{
		h_arr[i] = rand()%40;
		printf("%d ",h_arr[i]);
	}	
	
	oddEvenTranspositionSort(h_arr, n);
	
	printf("\nSorted Array: ");
	for (int i = 0; i < n; i++)
		printf("%d ", h_arr[i]);

	printf("\n");
}
