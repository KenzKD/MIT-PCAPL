#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__ __device__ void printMatrix(const char * string, int * A, int width) 
{
  printf("%s\n", string);
  for (int i = 0; i < width; i++) 
  {
    for (int j = 0; j < width; j++)
      printf("%d,\t ", A[i * width + j]);
    printf("\n");
  }
  printf("\n");
}

__host__ void clearMatrix(int * A, int width) 
{
  for (int i = 0; i < width; i++)
    for (int j = 0; j < width; j++)
      A[i * width + j] = 0;
}

__global__ void MatMulti_2a(int * A, int * B, int * C, int width) 
{
  int row = threadIdx.y;
  int k = 0;
  for (int i = 0; i < width; i++) 
  {
    k = 0;
    for (int j = 0; j < width; j++)
      k += A[row * width + j] * B[i + width * j];
    C[row * width + i] = k;
  }
}

__global__ void MatMulti_2b(int * A, int * B, int * C, int width) 
{
  int col = threadIdx.x;
  int k = 0;
  for (int i = 0; i < width; i++) 
  {
    k = 0;
    for (int j = 0; j < width; j++)
      k += A[i * width + j] * B[col + j * width];
    C[i * width + col] = k;
  }
}

__global__ void MatMulti_2c(int * A, int * B, int * C, int width) 
{
  int col = threadIdx.x;
  int row = threadIdx.y;
  int k = 0;
  for (int i = 0; i < width; i++)
    k += A[row * width + i] * B[col + i * width];
  C[row * width + col] = k;
}

__global__ void MatMulti_2d(int * A, int * B, int * C, int width) 
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  C[row * width + col] = 0;
  //calculating one element
  for (int k = 0; k < width; k++)
    C[row*width+ col] += A[row * width + k] * B[k * width + col];
}

void MatMulti(int * h_A, int * h_B, int * h_C, int width) 
{
  int * d_A, * d_B, * d_C;
  int size = width*width*sizeof(int);
  
  hipMalloc((void ** ) & d_A, size);
  hipMalloc((void ** ) & d_B, size);
  hipMalloc((void ** ) & d_C, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  
  dim3 dimBlock(1, 1, 1);
  dim3 dimGrid(1, 1, 1);
  
  dimBlock.x = 1;
  dimBlock.y = width;
  dimBlock.z = 1;
  MatMulti_2a <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2a kernel): ", h_C, width);
  clearMatrix(h_C, width);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  dimBlock.x = width;
  dimBlock.y = 1;
  dimBlock.z = 1;
  MatMulti_2b <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2b kernel): ", h_C, width);
  clearMatrix(h_C, width);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

  dimBlock.x = width;
  dimBlock.y = width;
  dimBlock.z = 1;
  MatMulti_2c <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2c kernel): ", h_C, width);

  dimBlock.x = 2;
  dimBlock.y = 2;
  dimBlock.z = 1;
  dimGrid.x = 2;
  dimGrid.y = 2;
  dimGrid.z = 1;
  MatMulti_2d <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, width);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  printMatrix("A*B: (from 2d kernel): ", h_C, width);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() 
{
  int * A, * B, * C, width = 3;
  A = (int * ) calloc(width * width, sizeof(int));
  B = (int * ) calloc(width * width, sizeof(int));
  C = (int * ) calloc(width * width, sizeof(int));
  
  for (int i = 0; i < width; i++) 
  {
    for (int j = 0; j < width; j++) 
    {
      A[i * width + j] = rand() % 10;
      B[i * width + j] = rand() % 11;
    }
  }
  printMatrix("A:", A, width);
  printMatrix("B:", B, width);
  MatMulti(A, B, C, width);
  return 0;
}