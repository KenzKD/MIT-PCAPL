#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void word_count_kernel(char* str, char* key, int* word_indices, int* result, int* key_len)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int si = word_indices[idx];
	int ei = word_indices[idx+1];
	char word[100];
	int i = 0;
	int corr_count=0;

	for (i = 0; i < (ei-si-1); i++)
	{
		word[i] = str[si+1+i];
	}

	word[i] = '\0';

	int i1 = 0;
	int i2 = 0;
	int is_equal = 0;

	while (word[i1] != '\0' && key[i2] != '\0')
	{
		if (word[i1] == key[i2])
		{
			i1++;
			i2++;
			corr_count++;
		}
		else
		{
			is_equal = 0;
			break;
		}
	}

	if(corr_count==(*key_len))
	{
		is_equal=1;
	}

	if (is_equal == 1)
	{
		atomicAdd(result, 1);
	}
}

int main()
{
	char str[100] = " apple banana mango apple laptop apple mango banana laptop confuse ";
	char key[100] = "apple";

	int str_len = strlen(str);
	int key_len = strlen(key);
	int word_count = 0;

	for (int i = 0; i < str_len; i++)
	{
		if (str[i] == ' ')
		{
			word_count++;
		}
	}

	int* word_indices;
	int wi = -1;

	word_indices = (int*) (malloc(word_count * sizeof(int)));

	for (int i = 0; i < str_len; i++)
	{
		if (str[i] == ' ')
		{
			word_indices[++wi] = i;
		}
	}

	int result = 0;

	char* d_str;
	char* d_key;
	int* d_word_indices;
	int* d_result;
	int* d_keylen;

	hipMalloc((void**)&d_str, str_len * sizeof(char));
	hipMalloc((void**)&d_key, key_len * sizeof(char));
	hipMalloc((void**)&d_word_indices, (word_count+1) * sizeof(int));
	hipMalloc((void**)&d_result, sizeof(int));
	hipMalloc((void**)&d_keylen, sizeof(int));

	hipMemcpy(d_str, str, str_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_key, key, key_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_word_indices, word_indices, (word_count+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_result, &result, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_keylen, &key_len, sizeof(int), hipMemcpyHostToDevice);


	word_count_kernel<<<1, word_count>>>(d_str, d_key, d_word_indices, d_result,d_keylen);

	hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

	printf("Input String: %s\n", str);
	printf("Key: %s\n", key);
	if(result!=1)
	{
		printf("Total occurrences of %s is %d\n", key, result);
	}
	else
	{
		printf("Total occurrences of %s is %d\n", key, result);
	}

	hipFree(d_str);
	hipFree(d_key);
	hipFree(d_result);

	return 0;
}